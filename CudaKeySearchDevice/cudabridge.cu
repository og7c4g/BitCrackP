#include "hip/hip_runtime.h"
#include "cudabridge.h"


__global__ void keyFinderKernel(int points, int compression);
__global__ void keyFinderKernelWithDouble(int points, int compression);

void callKeyFinderKernel(int blocks, int threads, int points, bool useDouble, int compression)
{
	if(useDouble) {
		keyFinderKernelWithDouble <<<blocks, threads >>>(points, compression);
	} else {
		keyFinderKernel <<<blocks, threads>>> (points, compression);
	}
	waitForKernel();
}


void waitForKernel()
{
    // Check for kernel launch error
    hipError_t err = hipGetLastError();

    if(err != hipSuccess) {
        throw cuda::CudaException(err);
    }
 
    // Wait for kernel to complete
    err = hipDeviceSynchronize();
	fflush(stdout);
	if(err != hipSuccess) {
		throw cuda::CudaException(err);
	}
}